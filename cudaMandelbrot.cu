#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define grid 10
#define block 1024
bool InitCUDA()
{
    int count;
    hipGetDeviceCount(&count);
    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }
    int i;
    for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                break;
            }
        }
    }
    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }
    hipSetDevice(i);
    return true;
}

struct complex{
float r;
float i;
};

__device__ float mod(struct complex a){
return sqrt(a.r*a.r+a.i*a.i);
}

__device__ struct complex add(struct complex a,struct complex b){
struct complex c;
c.r=a.r+b.r;
c.i=a.i+b.i;
return c;
}
__device__ struct complex square(struct complex a){
struct complex b;
b.r=a.r*a.r-a.i*a.i;
b.i=2*a.r*a.i;
return b;
}
#define maxx 1000
__global__ void man(struct complex *p,int *o,int tol,int pixels)
{
int id=blockIdx.x*blockDim.x+threadIdx.x;
int i,j,k;
struct complex z,c;
	for(i=0;i*tol+id<pixels;i++){
		z.r=0;
		z.i=0;
		c=p[i*tol+id];
		k=maxx;
		for(j=1;j<maxx;j++){
			z=add(square(z),c);
			if((k==maxx)&&(mod(z)>2.0f)){
				k=j;
			}

		}
		o[i*tol+id]=k;
	}
}



#define height 2000
#define width 3000
#define xstart -2.0f
#define xend 1.0f
#define ystart -1.0f
#define yend 1.0f
int main()
{
    if (!InitCUDA()) {
        return 0;
    }
	struct complex* pic=(struct complex*)malloc(sizeof(struct complex)*width*height);
	int i,j;
	for(i=0;i<width;i++){
		for(j=0;j<height;j++){
			pic[i*height+j].r=xstart+(xend-xstart)*i/width;
			pic[i*height+j].i=ystart+(yend-ystart)*j/height;
		}
	}
	struct complex* picd;
	int *o=(int*)malloc(sizeof(int)*width*height); 
	int *od;
	hipMalloc(&picd,sizeof(struct complex)*width*height);
	hipMemcpy(picd,pic,sizeof(struct complex)*width*height,hipMemcpyHostToDevice);
	
	hipMalloc(&od,sizeof(int)*width*height);
	man<<<grid,block>>>(picd,od,grid*block,height*width);
	hipMemcpy(o,od,sizeof(int)*width*height,hipMemcpyDeviceToHost);
	FILE *fp=fopen("./out.ppm","w");
	fprintf(fp,"P3\n%d %d\n255\n",width,height);
	int r,g,b,col;
	for(i=0;i<height;i++){
		for(j=0;j<width;j++){
			col=o[j*height+i];
			if(col==maxx){
			r=0;
			g=0;
			b=0;
			}else{
			b=(int)max(min((0+(int)(512.0f/sqrt(o[j*height+i]))),255),0);
			g=0;
			r=0;
			}
			fprintf(fp,"%d %d %d ",r,g,b);
		}
		fprintf(fp,"\n");
	}
	fclose(fp);
	
	free(pic);
	free(o);
	hipFree(picd);
	hipFree(od);
	hipDeviceReset();
    return 0;
}
